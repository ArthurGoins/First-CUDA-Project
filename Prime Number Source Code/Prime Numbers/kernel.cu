#include "hip/hip_runtime.h"
/*==============================================================================================*
* Project: CUDA Bemchmark based op calculating prime numbers									*
* Developed with Visual Studio 2015 and CUDA Toolkit 8.0										*
* Written by:	Arthur Goins																	*
*				Computer Engineering student at NCSU											*
*				ajgoins@ncsu.edu																*
*																								*
* Function:		To test and quantify the compute performance of a multi-threaded CUDA device	*
*===============================================================================================*/

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <conio.h>
#include <time.h>

#define MAXARRAY 20000000											//calculate primes until this number
#define THREADS 100													//each thread will check 200,000 numbers (each thread will check every 100 integers)

__global__ 
void checkPrime()													//CUDA function
{
	int flag = 0;													//flag to identify how the following for loop exited
	int loop = 1;													//variable to allow the function to exit the while loop
	int x = threadIdx.x;											//save the thread ID as an integer (henceforth referred to as the 'thread integer' and it is the number that will be tested)
	while (loop)
	{
		for (int i = 2; i <= (x / 2); ++i)							//set the integer i to every integer value between 2 and half of the previously saved thread integer
		{		
			if ((x%i) == 0)											//check if the thread integer is evenly divisible by i
			{
				flag = 1;											//the thread integer is not prime, so 'flag' is set to indicate this
				break;												//exit the for loop
			}

		}
		if (flag == 0) {											//if 'flag' is still 0, the thread integer is prime
			x += THREADS;											//increment the thread integer by the macro defined earlier
		}
		else
		{
			x += THREADS;											//increment the thread integer by the macro defined earlier
		}
		if (x >= MAXARRAY)											//once a thread reaches the target number, stop its loop
			loop = 0;
	}
}

int main()
{
	clock_t start, end;												//timing variables
	double cpu_time;											
	double time_per_cycle;

	const int cycles = 150;											//variables for setting the number of cycles to run
	int iteration = 0;

	dim3 dimBlock(1000, 1);											//block size and thread configuration
	dim3 dimGrid(THREADS, 1);

	printf("starting test...\n");

	checkPrime << <dimGrid, dimBlock >> > ();						//run the function a few times to allow the GPU to warm up
	hipDeviceSynchronize();
	checkPrime << <dimGrid, dimBlock >> > ();
	hipDeviceSynchronize();
	checkPrime << <dimGrid, dimBlock >> > ();
	hipDeviceSynchronize();

	start = clock();												//get the current time
	checkPrime << <dimGrid, dimBlock >> > ();						//run the CUDA function
	hipDeviceSynchronize();										//wait for the function to finish
	end = clock();													//get the current time
	cpu_time = ((double)(end - start)) / CLOCKS_PER_SEC;			//determine the time per cycle
	time_per_cycle = cpu_time;										//store the fist time in the accumulation variable
	
	while (iteration < cycles)										//loop the same cycle as above 150 times
	{
		start = clock();
		checkPrime << <dimGrid, dimBlock >> > ();
		hipDeviceSynchronize();
		end = clock();
		cpu_time = ((double)(end - start)) / CLOCKS_PER_SEC;
		time_per_cycle = (time_per_cycle + cpu_time) / 2;
		iteration++;
		printf("Time: %f \t %d out of 150\n", cpu_time, iteration);	//print each time
	}

	printf("Average time per cycle: %f\n", time_per_cycle);			//print average time
	printf("Score: %f\n", 1000 / time_per_cycle);					//print score
	printf("Comparisons:\n");
	printf("GTX 1070:\t 17418\n");
	printf("GTX 960:\t 7016\n");
	printf("GTX 680:\t 6192\n");
	printf("GTX 650Ti:\t 2943\n");
	printf("GTX 960M:\t 2628\n");
	system("pause");
	return 0;
}